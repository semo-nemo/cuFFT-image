#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "helper/helper_functions.h"
#include "helper/hip/hip_runtime_api.h"

#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hipfft/hipfft.h>
#include <fstream>

using namespace std;
typedef float2 Complex;
#define rows 20 //1936/100;
#define columns 25 // 2592/100;

//Found at http://techqa.info/programming/question/36889333/cuda-cufft-2d-example


__global__ void filter_fft(Complex *a)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int column_i  = i % columns;
    int row_i = i % rows;
    // remove low frequency
    if (column_i < 5 || row_i < 5) {
        a[i].x = 0;
        a[i].y = 0;
    }
    // remove high frequency
    if (column_i > columns - 5 || row_i > rows - 5) {
        a[i].x = 0;
        a[i].y = 0;
    }
}


int main()
{ 
    int N = 5;

    int SIZE = rows * columns;


    Complex *fg = new Complex[SIZE];
    for (int i = 0; i < SIZE; i++){
        fg[i].x = 1;
        fg[i].y = 0;
    } 

    int mem_size = sizeof(Complex)* SIZE;

    hipfftComplex *d_signal;
    checkCudaErrors(hipMalloc((void **)&d_signal, mem_size)); 
    checkCudaErrors(hipMemcpy(d_signal, fg, mem_size, hipMemcpyHostToDevice));
 
    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan2d(&plan, rows, columns, HIPFFT_C2C);

    // Transform signal and filter
    printf("Transforming signal hipfftExecR2C\n");
    int direction = HIPFFT_FORWARD;
    hipfftResult res;
    res = hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, (int)direction); 
    
    printf("Filter some FFT components<<< >>>\n");
    filter_fft <<< N, N >> >(d_signal); 

    // Transform signal back
    printf("Transforming signal back hipfftExecC2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, (int)HIPFFT_BACKWARD);

    Complex *result = new Complex[SIZE];
    hipMemcpy(result, d_signal, sizeof(Complex)*SIZE, hipMemcpyDeviceToHost);

 

    delete result, fg;
    hipfftDestroy((hipfftHandle)plan);
    hipFree(d_signal); 

}